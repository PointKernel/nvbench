#include "hip/hip_runtime.h"
/*
 *  Copyright 2020 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/detail/measure_cold.cuh>

#include <nvbench/benchmark_base.cuh>
#include <nvbench/device_info.cuh>
#include <nvbench/printer_base.cuh>
#include <nvbench/state.cuh>
#include <nvbench/summary.cuh>

#include <nvbench/detail/statistics.cuh>

#include <fmt/format.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <stdexcept>
#include <variant>

namespace nvbench::detail
{

measure_cold_base::measure_cold_base(state &exec_state)
    : m_state{exec_state}
    , m_min_samples{exec_state.get_min_samples()}
    , m_max_noise{exec_state.get_max_noise()}
    , m_min_time{exec_state.get_min_time()}
    , m_skip_time{exec_state.get_skip_time()}
    , m_timeout{exec_state.get_timeout()}
{}

void measure_cold_base::check()
{
  const auto device = m_state.get_device();
  if (!device)
  {
    throw std::runtime_error(fmt::format("{}:{}: Device required for `cold` "
                                         "measurement.",
                                         __FILE__,
                                         __LINE__));
  }
  if (!device->is_active())
  { // This means something went wrong higher up. Throw an error.
    throw std::runtime_error(fmt::format("{}:{}: Internal error: Current "
                                         "device is not active.",
                                         __FILE__,
                                         __LINE__));
  }
}

void measure_cold_base::generate_summaries()
{
  const auto d_samples = static_cast<double>(m_total_samples);
  {
    auto &summ = m_state.add_summary("Number of Samples (Cold)");
    summ.set_string("hint", "sample_size");
    summ.set_string("short_name", "Samples");
    summ.set_string("description",
                    "Number of kernel executions in cold time measurements.");
    summ.set_int64("value", m_total_samples);
  }

  const auto avg_cpu_time = m_total_cpu_time / d_samples;
  {
    auto &summ = m_state.add_summary("Average CPU Time (Cold)");
    summ.set_string("hint", "duration");
    summ.set_string("short_name", "CPU Time");
    summ.set_string("description",
                    "Average isolated kernel execution time observed "
                    "from host.");
    summ.set_float64("value", avg_cpu_time);
  }

  {
    auto &summ = m_state.add_summary("CPU Relative Standard Deviation (Cold)");
    summ.set_string("hint", "percentage");
    summ.set_string("short_name", "Noise");
    summ.set_string("description",
                    "Relative standard deviation of the cold CPU execution "
                    "time measurements.");
    summ.set_float64("value", m_cpu_noise);
  }

  const auto avg_cuda_time = m_total_cuda_time / d_samples;
  {
    auto &summ = m_state.add_summary("Average GPU Time (Cold)");
    summ.set_string("hint", "duration");
    summ.set_string("short_name", "GPU Time");
    summ.set_string("description",
                    "Average isolated kernel execution time as measured "
                    "by CUDA events.");
    summ.set_float64("value", avg_cuda_time);
  }

  {
    auto &summ = m_state.add_summary("GPU Relative Standard Deviation (Cold)");
    summ.set_string("hint", "percentage");
    summ.set_string("short_name", "Noise");
    summ.set_string("description",
                    "Relative standard deviation of the cold GPU execution "
                    "time measurements.");
    summ.set_float64("value", m_cuda_noise);
  }

  if (!m_cuda_times.empty())
  { // Percentiles and histogram:
    const auto percentiles = std::vector<int>{0, 1, 25, 50, 75, 99, 100};
    std::sort(m_cuda_times.begin(), m_cuda_times.end());
    const auto results = nvbench::detail::compute_percentiles(m_cuda_times,
                                                              percentiles);

    const auto histo_min    = results[1]; // 1st percentile
    const auto histo_max    = results[5]; // 99th percentile
    const auto histo_bins   = std::size_t{50};
    const auto histo_stride = (histo_max - histo_min) /
                              static_cast<double>(histo_bins);

    const auto histo = nvbench::detail::compute_histogram(m_cuda_times,
                                                          histo_min,
                                                          histo_stride,
                                                          histo_bins);
    {
      auto &summ = m_state.add_summary("GPU Time Percentiles (Cold)");
      summ.set_string("hide", "Non-standard format.");
      summ.set_string("hint", "percentiles");
      assert(percentiles.size() == results.size());
      for (std::size_t i = 0; i < results.size(); ++i)
      {
        fmt::print("Percentile {:<3}: {}\n", percentiles[i], results[i]);
        summ.set_float64(fmt::format("{}", percentiles[i]), results[i]);
      }
    }

    {
      auto &summ = m_state.add_summary("GPU Time Histogram (Cold)");
      summ.set_string("hide", "Non-standard format.");
      summ.set_string("hint", "histogram");
      summ.set_float64("min", histo_min);
      summ.set_float64("stride", histo_stride);
      summ.set_int64("bins", static_cast<nvbench::int64_t>(histo_bins));
      assert(histo.size() == histo_bins + 2);
      const auto scale =
        std::reduce(histo.cbegin(), histo.cend(), 0, [](auto a, auto b) {
          return a > b ? a : b;
        });
      for (std::size_t i = 0; i < histo_bins + 2; ++i)
      {
        fmt::print("Bin {:<2}: {:5} |{:0>{}}\n",
                   i,
                   histo[i],
                   "|",
                   (histo[i] * 50) / scale);
        summ.set_int64(fmt::format("bin {}", i), histo[i]);
      }
    }
  }

  if (const auto items = m_state.get_element_count(); items != 0)
  {
    auto &summ = m_state.add_summary("Element Throughput");
    summ.set_string("hint", "item_rate");
    summ.set_string("short_name", "Elem/s");
    summ.set_string("description",
                    "Number of input elements handled per second.");
    summ.set_float64("value", static_cast<double>(items) / avg_cuda_time);
  }

  if (const auto bytes = m_state.get_global_memory_rw_bytes(); bytes != 0)
  {
    const auto avg_used_gmem_bw = static_cast<double>(bytes) / avg_cuda_time;
    {
      auto &summ = m_state.add_summary("Average Global Memory Throughput");
      summ.set_string("hint", "byte_rate");
      summ.set_string("short_name", "GlobalMem BW");
      summ.set_string("description",
                      "Number of bytes read/written per second to the CUDA "
                      "device's global memory.");
      summ.set_float64("value", avg_used_gmem_bw);
    }

    {
      const auto peak_gmem_bw = static_cast<double>(
        m_state.get_device()->get_global_memory_bus_bandwidth());

      auto &summ = m_state.add_summary("Percent Peak Global Memory Throughput");
      summ.set_string("hint", "percentage");
      summ.set_string("short_name", "BWPeak");
      summ.set_string("description",
                      "Global device memory throughput as a percentage of the "
                      "device's peak bandwidth.");
      summ.set_float64("value", avg_used_gmem_bw / peak_gmem_bw * 100.);
    }
  }

  // Log if a printer exists:
  if (auto printer_opt_ref = m_state.get_benchmark().get_printer();
      printer_opt_ref.has_value())
  {
    auto &printer = printer_opt_ref.value().get();

    if (m_max_time_exceeded)
    {
      const auto timeout = m_timeout_timer.get_duration();

      if (m_cuda_noise > m_max_noise)
      {
        printer.log(nvbench::log_level::warn,
                    fmt::format("Current measurement timed out ({:0.2f}s) "
                                "while over noise threshold ({:0.2f}% > "
                                "{:0.2f}%)",
                                timeout,
                                m_cuda_noise,
                                m_max_noise));
      }
      if (m_total_samples < m_min_samples)
      {
        printer.log(nvbench::log_level::warn,
                    fmt::format("Current measurement timed out ({:0.2f}s) "
                                "before accumulating min_samples ({} < {})",
                                timeout,
                                m_total_samples,
                                m_min_samples));
      }
      if (m_total_cuda_time < m_min_time)
      {
        printer.log(nvbench::log_level::warn,
                    fmt::format("Current measurement timed out ({:0.2f}s) "
                                "before accumulating min_time ({:0.2f}s < "
                                "{:0.2f}s)",
                                timeout,
                                m_total_cuda_time,
                                m_min_time));
      }
    }

    // Log to stdout:
    printer.log(nvbench::log_level::pass,
                fmt::format("Cold: {:0.6f}ms GPU, {:0.6f}ms CPU, {:0.2f}s "
                            "total GPU, {}x",
                            avg_cuda_time * 1e3,
                            avg_cpu_time * 1e3,
                            m_total_cuda_time,
                            m_total_samples));
  }
}

void measure_cold_base::check_skip_time(nvbench::float64_t warmup_time)
{
  if (m_skip_time > 0. && warmup_time < m_skip_time)
  {
    auto reason = fmt::format("Warmup time did not meet skip_time limit: "
                              "{:0.3f}us < {:0.3f}us.",
                              warmup_time * 1e6,
                              m_skip_time * 1e6);

    m_state.skip(reason);
    throw std::runtime_error{std::move(reason)};
  }
}

} // namespace nvbench::detail
